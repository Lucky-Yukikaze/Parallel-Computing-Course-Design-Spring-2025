﻿#include <iostream>
#include <vector>
#include <string>
#include <cstdio>
#include <cstdlib>
#include <iomanip>
#include <cassert>
#include <chrono>
#include <io.h>
#include <omp.h>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>

using namespace cv;
using namespace std;

void rgb2grayOpenMP(unsigned char* const d_in, unsigned char* const d_out, int imgHeight, int imgWidth)
{
#pragma omp parallel for collapse(2)
    for (int i = 0; i < imgHeight; ++i)
    {
        for (int j = 0; j < imgWidth; ++j)
        {
            int idx = i * imgWidth + j;
            d_out[idx] = static_cast<unsigned char>(
                0.299f * d_in[idx * 3] +
                0.587f * d_in[idx * 3 + 1] +
                0.114f * d_in[idx * 3 + 2]);
        }
    }
}

void getFiles(string path, vector<string>& files)
{
    intptr_t hFile = 0;
    struct _finddata_t fileinfo;
    string p;
    if ((hFile = _findfirst(p.assign(path).append("\\*").c_str(), &fileinfo)) != -1)
    {
        do
        {
            if ((fileinfo.attrib & _A_SUBDIR))
            {
                if (strcmp(fileinfo.name, ".") != 0 && strcmp(fileinfo.name, "..") != 0)
                    getFiles(p.assign(path).append("\\").append(fileinfo.name), files);
            }
            else
            {
                files.push_back(p.assign(path).append("\\").append(fileinfo.name));
            }
        } while (_findnext(hFile, &fileinfo) == 0);
        _findclose(hFile);
    }
}

int OpenMPFunc(string inputfilename, double& openmpsumtime)
{
    Mat srcImg = imread(inputfilename);
    if (srcImg.empty())
    {
        cerr << "无法打开图片: " << inputfilename << endl;
        return -1;
    }

    int imgHeight = srcImg.rows;
    int imgWidth = srcImg.cols;

    Mat grayImg(imgHeight, imgWidth, CV_8UC1, Scalar(0));

    // OpenMP 并行部分
    auto omp_start = chrono::high_resolution_clock::now();
    rgb2grayOpenMP(srcImg.data, grayImg.data, imgHeight, imgWidth);
    auto omp_end = chrono::high_resolution_clock::now();

    double omp_time = chrono::duration<double>(omp_end - omp_start).count();
    openmpsumtime += omp_time;
    cout << fixed << setprecision(10) << "OpenMP执行时间: " << omp_time << " s" << endl;

    // 保存灰度图
    try
    {
        string outputname = "./GrayPicture/" + inputfilename.substr(10, inputfilename.length() - 14) + "_to_gray.png";
        vector<int> compression_params = { IMWRITE_PNG_COMPRESSION, 0 };
        imwrite(outputname, grayImg, compression_params);
        cout << "保存成功 " << outputname << endl;
    }
    catch (runtime_error& ex)
    {
        cerr << "图像保存失败: " << ex.what() << endl;
        return 1;
    }

    return 0;
}

int main()
{
    string filePath = "./Picture";
    vector<string> files;
    getFiles(filePath, files);

    int size = files.size();
    cout << "图片数量：" << size << endl;

    // 用户输入线程数
    int thread_num;
    cout << "请输入使用的线程数：";
    cin >> thread_num;
    omp_set_num_threads(thread_num);

    double openmpsumtime = 0;

    for (int i = 0; i < size; i++)
    {
        cout << "处理第 " << i + 1 << " 张图片：" << files[i] << endl;
        OpenMPFunc(files[i], openmpsumtime);
        cout << endl;
    }

    cout << "总计 OpenMP 执行时间: " << openmpsumtime << " s" << endl;

    FILE* fp = fopen("time.txt", "a");
    if (fp)
    {
        fprintf(fp, "线程数：%d，OpenMP总时间：%.10lf s\n", thread_num, openmpsumtime);
        fclose(fp);
    }

    return 0;
}