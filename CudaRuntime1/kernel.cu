#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <string>
#include <cassert>
#include <vector>
#include <math.h>
#include <time.h>
#include <io.h>
#include <chrono>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>
#include "opencv2/highgui.hpp" 
#include "opencv2/imgcodecs/legacy/constants_c.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace cv;
using namespace std;

#define THREAD_NUM 32


//输入图像为BGR图，将其转化为gray图
__global__ void rgb2grayInCuda(uchar3* dataIn, unsigned char* dataOut, int imgHeight, int imgWidth)
{
	//图片二维扫描，分别有x方向，y方向的像素点
	int xIndex = threadIdx.x + blockIdx.x * blockDim.x;	//表示x方向上的ID
	int yIndex = threadIdx.y + blockIdx.y * blockDim.y;	//表示y方向上的ID
	//灰度变换操作
	if (xIndex < imgWidth && yIndex < imgHeight)
	{
		uchar3 rgb = dataIn[yIndex * imgWidth + xIndex];
		dataOut[yIndex * imgWidth + xIndex] = 0.299f * rgb.x + 0.587f * rgb.y + 0.114f * rgb.z;
	}
}
//串行转换灰度图像
void rgb2grayincpu(unsigned char* const d_in, unsigned char* const d_out, uint imgheight, uint imgwidth)
{
	//使用两重循环嵌套实现x方向 y方向的变换
	for (int i = 0; i < imgheight; i++)
	{
		for (int j = 0; j < imgwidth; j++)
		{
			d_out[i * imgwidth + j] = 0.299f * d_in[(i * imgwidth + j) * 3]
				+ 0.587f * d_in[(i * imgwidth + j) * 3 + 1]
				+ 0.114f * d_in[(i * imgwidth + j) * 3 + 2];
		}
	}
}


int CUDAfunc(string inputfilename, double& gpusumtime, double& cpusumtime) {
	//传入图片
	Mat srcImg = imread(inputfilename);
	FILE* fp;//创建运行时间文件

	//读取图片像素值
	int imgHeight = srcImg.rows;
	int imgWidth = srcImg.cols;

	Mat grayImg(imgHeight, imgWidth, CV_8UC1, Scalar(0));	//输出灰度图

	//在GPU中开辟输入输出空间
	uchar3* d_in;
	unsigned char* d_out;
	int* d_hist;

	//分配内存空间
	hipMalloc((void**)&d_in, imgHeight * imgWidth * sizeof(uchar3));
	hipMalloc((void**)&d_out, imgHeight * imgWidth * sizeof(unsigned char));
	hipMalloc((void**)&d_hist, 256 * sizeof(int));

	//将图像数据传入GPU中
	hipMemcpy(d_in, srcImg.data, imgHeight * imgWidth * sizeof(uchar3), hipMemcpyHostToDevice);

	dim3 threadsPerBlock(THREAD_NUM, THREAD_NUM);
	dim3 blocksPerGrid((imgWidth + threadsPerBlock.x - 1) / threadsPerBlock.x, (imgHeight + threadsPerBlock.y - 1) / threadsPerBlock.y);
	//cuda灰度化
	//计时开始
	auto gpustart = chrono::system_clock::now();
	//调用核函数
	rgb2grayInCuda << <blocksPerGrid, threadsPerBlock >> > (d_in, d_out, imgHeight, imgWidth);
	//同步CPU和gpu，否则测速结果为cpu启动内核函数的速度
	hipDeviceSynchronize();
	//计时结束
	auto gpuend = chrono::system_clock::now();
	//计算时间差
	auto gpuduration = chrono::duration_cast<chrono::microseconds>(gpuend - gpustart);
	double gput = gpuduration.count();
	//微秒转化为秒
	double gputime = gput / 1000000;
	gpusumtime += gputime;
	//打印cuda并行执行时间
	cout << setiosflags(ios::fixed) << setprecision(10) << "CUDA执行时间： " << gputime << " s" << endl;
	//将数据从GPU传回CPU
	hipMemcpy(grayImg.data, d_out, imgHeight * imgWidth * sizeof(unsigned char), hipMemcpyDeviceToHost);
	vector<int> compression_params;
	compression_params.push_back(CV_IMWRITE_PNG_COMPRESSION);
	compression_params.push_back(0);
	//释放内存
	hipFree(d_in);
	hipFree(d_out);
	hipFree(d_hist);

	/*CPU串行开始*/
	//串行灰度化
	//计时开始
	auto cpustart = chrono::system_clock::now();
	//调用主函数
	rgb2grayincpu(srcImg.data, grayImg.data, imgHeight, imgWidth);
	//计时结束
	auto cpuend = chrono::system_clock::now();
	//计算时间差
	auto cpuduration = chrono::duration_cast<chrono::microseconds>(cpuend - cpustart);
	double cput = cpuduration.count();
	//微秒转化为秒
	double cputime = cput / 1000000;
	cpusumtime += cputime;
	//打印串行执行时间
	cout << setiosflags(ios::fixed) << setprecision(10) << "CPU执行时间： " << cputime << " s" << endl;

	/*输出灰度图片*/
	try
	{
		int len = inputfilename.length();
		cout << "inputfilename.length:" << len << endl;
		string str = "./GrayPicture/";
		imwrite(str + inputfilename.substr(10, len - 14) + "_to_gray.png", grayImg, compression_params);
		cout << str + inputfilename.substr(10, len - 14) + "_to_gray.png" << endl;

		//在GrayPicture文件夹中，生成灰度变换后的结果图片  
	}
	catch (runtime_error& ex)
	{
		fprintf(stderr, "图像转换成PNG格式发生错误：%s\n", ex.what());
		return 1;
	}
	return 0;
}

//批量读取图片
void getFiles(string path, vector<string>& files)
{
	//文件句柄  
	intptr_t hFile = 0;
	//文件信息  
	struct _finddata_t fileinfo;
	string p;
	if ((hFile = _findfirst(p.assign(path).append("\\*").c_str(), &fileinfo)) != -1)
	{
		do
		{
			//如果是目录,迭代之  
			//如果不是,加入列表  
			if ((fileinfo.attrib & _A_SUBDIR))
			{
				if (strcmp(fileinfo.name, ".") != 0 && strcmp(fileinfo.name, "..") != 0)
					getFiles(p.assign(path).append("\\").append(fileinfo.name), files);
			}
			else
			{
				files.push_back(p.assign(path).append("\\").append(fileinfo.name));
			}
		} while (_findnext(hFile, &fileinfo) == 0);
		_findclose(hFile);
	}
}
int main()
{
	//图片文件路径，在项目文件下的Picture文件夹里面
	string filePath = "./Picture";
	vector<string> files;
	//读取图片文件
	getFiles(filePath, files);
	//读取图片数量
	int size = files.size();
	//输出图片数量
	cout << "图片数量：" << size << endl;

	double gpusumtime = 0, cpusumtime = 0;
	for (int i = 0; i < size; i++)
	{
		cout << "第 " << i + 1 << "/" << size << " 张图片" << endl;
		cout << files[i].c_str() << endl;
		CUDAfunc(files[i].c_str(), gpusumtime, cpusumtime);
		cout << endl;
	}

	cout << "总GPU时间：" << gpusumtime << " s" << "\n" << "总CPU时间：" << cpusumtime << " s" << endl;
	FILE* fp;
	fp = fopen("time.txt", "a");
	fprintf(fp, "总CPU时间： %.10lf s ,总GPU时间： %.10lf s \n", cpusumtime, gpusumtime);
	fclose(fp);

	return 0;
}